// Created by Kaustubh Shivdikar
//
//  (C) All Rights Reserved


# include "../lib/functions.h"
# include "../lib/debugger.h"
# include "hip/hip_runtime.h"
using namespace std;

// Global Variables





// ************ MAIN FUNCTION ************


int main (int argc, char *argv[]) {

		Input input;
		get_inputs (argc, argv, input, 1);
		hipDeviceSynchronize();


		cout << "\n\e[1;31mProgram End\e[0m\n\n\n";
		return 0;
}

